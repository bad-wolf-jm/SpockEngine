/// @file   KernelLaunchers.cu
///
/// @brief  C++ API for Cuda computation launchers
///
/// @author Jean-Martin Albert
///
/// @copyright (c) 2023 Jean-Martin Albert. All rights reserved.

#include "KernelLaunchers.h"

#include <chrono>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdexcept>
#include <variant>

#include "Core/Logging.h"

#include "HelperMacros.h"

#include "DeviceKernels.inl"

namespace SE::TensorOps
{

    struct RandomNumberGenerator
    {
        hiprandGenerator_t Generator = nullptr;

        RandomNumberGenerator()
        {
            auto lNow   = std::chrono::system_clock::now();
            auto lNowNS = std::chrono::time_point_cast<std::chrono::nanoseconds>( lNow );
            auto lValue = lNowNS.time_since_epoch();
            CURAND_ASSERT( hiprandCreateGenerator( &Generator, HIPRAND_RNG_PSEUDO_DEFAULT ) );
            CURAND_ASSERT( hiprandSetPseudoRandomGeneratorSeed( Generator, lValue.count() ) );
        }

        ~RandomNumberGenerator()
        {
            hiprandDestroyGenerator( Generator );
        }
    };

    template <typename _Ty>
    static void ConstantFillImpl( MultiTensor &aArray, scalar_value_t &aConstant )
    {
        int lBlockCount = ( aArray.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aArray.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::ConstantFill<_Ty><<<lGridDim, lBlockDim>>>( aArray, std::get<_Ty>( aConstant ) );
    }

    template <typename _Ty>
    static void ConstantFillImpl( MultiTensor &aArray, MemoryBuffer &aInitialValues )
    {
        int lBlockCount = ( aArray.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aArray.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::ConstantFill<_Ty><<<lGridDim, lBlockDim>>>( aArray, aInitialValues );
    }

    void ConstantFill( scalar_type_t aTensorElementType, MultiTensor &aArray, MemoryBuffer &aInitialValues )
    {
        DISPATCH_BY_TYPE( aTensorElementType, ConstantFillImpl, ( aArray, aInitialValues ) );
    }

    void ConstantFill( scalar_type_t aTensorElementType, MultiTensor &aArray, scalar_value_t &aInitialValues )
    {
        DISPATCH_BY_TYPE( aTensorElementType, ConstantFillImpl, ( aArray, aInitialValues ) );
    }

    void RandomUniformFill( scalar_type_t aTensorElementType, MultiTensor &aArray )
    {
        switch( aTensorElementType )
        {
        case scalar_type_t::FLOAT32:
        {
            RandomNumberGenerator lGenerator{};
            CURAND_ASSERT( hiprandGenerateUniform( lGenerator.Generator, aArray.DataAs<float>(), aArray.SizeAs<float>() ) );
        }
        break;
        case scalar_type_t::FLOAT64:
        {
            RandomNumberGenerator lGenerator{};
            CURAND_ASSERT( hiprandGenerateUniformDouble( lGenerator.Generator, aArray.DataAs<double>(), aArray.SizeAs<double>() ) );
        }
        break;
        default:
            std::runtime_error( "Random number type can only be float or double" );
        }
    }

    void RandomNormalFill( scalar_type_t aTensorElementType, MultiTensor &aArray, scalar_value_t &aMu, scalar_value_t &aSigma )
    {
        switch( aTensorElementType )
        {
        case scalar_type_t::FLOAT32:
        {
            float lMean = std::get<float>( aMu );
            float lStd  = std::get<float>( aSigma );
            if( lStd <= 0.0f )
                std::runtime_error( "Variance parameter should be strictly positive" );
            RandomNumberGenerator lGenerator{};
            CURAND_ASSERT( hiprandGenerateNormal( lGenerator.Generator, aArray.DataAs<float>(), aArray.SizeAs<float>(), lMean, lStd ) );
        }
        break;
        case scalar_type_t::FLOAT64:
        {
            double lMean = std::get<double>( aMu );
            double lStd  = std::get<double>( aSigma );
            if( lStd <= 0.0f )
                std::runtime_error( "Variance parameter should be strictly positive" );
            RandomNumberGenerator lGenerator{};
            CURAND_ASSERT(
                hiprandGenerateNormalDouble( lGenerator.Generator, aArray.DataAs<double>(), aArray.SizeAs<double>(), lMean, lStd ) );
        }
        break;
        default:
            std::runtime_error( "Random number type can only be float or double" );
        }
    }

    template <typename _Ty>
    static void ARangeOpImpl( MultiTensor &aOut, MemoryBuffer &aLeft, MemoryBuffer &aRight, MemoryBuffer &aDelta,
                              uint32_t aMaxSubdivisions )
    {
        int lBlockCount = ( aMaxSubdivisions / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aOut.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::ARange<_Ty><<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight, aDelta );
    }

    void ARangeOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MemoryBuffer &aLeft, MemoryBuffer &aRight, MemoryBuffer &aDelta,
                   uint32_t aMaxSubdivisions )
    {
        switch( aTensorElementType )
        {
        case scalar_type_t::FLOAT32:
        {
            ARangeOpImpl<float>( aOut, aLeft, aRight, aDelta, aMaxSubdivisions );
            break;
        }
        case scalar_type_t::FLOAT64:
        {
            ARangeOpImpl<double>( aOut, aLeft, aRight, aDelta, aMaxSubdivisions );
            break;
        }
        default:
            throw std::runtime_error( "Linear space only supports float and double values" );
        }
    }

    template <typename _Ty>
    static void AddArrayToArrayImpl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AddArrayToArray<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    template <typename _Ty>
    static void AddArrayToArrayImpl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant, eBroadcastHint aBroadcastHint,
                                     MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                                     uint32_t aMaxBroadcastSizes )
    {
        int lBlockCount = ( aMaxBroadcastSizes / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), aMaxBlockSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AddArrayToArray<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant, aBroadcastHint, aBlockSizes, aBroadcastSizes );
    }

    template <typename _ScalarType>
    static void AddScalarToArrayImpl( MultiTensor &aOut, MultiTensor &aArray, scalar_value_t &aConstant )
    {
        int lBlockCount = ( aArray.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aArray.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AddScalarToArray<_ScalarType><<<lGridDim, lBlockDim>>>( aOut, aArray, std::get<_ScalarType>( aConstant ) );
    }

    template <typename _Ty>
    static void AddArrayToVectorImpl( MultiTensor &aOut, MultiTensor &aIn, MemoryBuffer &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AddArrayToVector<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    void AddOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, AddArrayToArrayImpl, ( aOut, aLeft, aRight ) );
    }

    void AddOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight,
                eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                uint32_t aMaxBroadcastSizes )
    {
        DISPATCH_BY_TYPE( aTensorElementType, AddArrayToArrayImpl,
                          ( aOut, aLeft, aRight, aBroadcastHint, aBlockSizes, aMaxBlockSize, aBroadcastSizes, aMaxBroadcastSizes ) );
    }

    void AddOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, AddScalarToArrayImpl, ( aOut, aLeft, aRight ) );
    }

    void AddOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, AddArrayToVectorImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _ScalarType>
    void MultiplyArrayByScalarImpl( MultiTensor &aOut, MultiTensor &aIn, scalar_value_t &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::MultiplyScalarByArray<_ScalarType><<<lGridDim, lBlockDim>>>( aOut, aIn, std::get<_ScalarType>( aConstant ) );
    }

    template <typename _Ty>
    static void MultiplyArrayByArrayImpl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::MultiplyArrayByArray<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    template <typename _Ty>
    static void MultiplyArrayByArrayImpl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant, eBroadcastHint aBroadcastHint,
                                          MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                                          uint32_t aMaxBroadcastSizes )
    {
        int lBlockCount = ( aMaxBroadcastSizes / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), aMaxBlockSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::MultiplyArrayByArray<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant, aBroadcastHint, aBlockSizes, aBroadcastSizes );
    }

    template <typename _Ty>
    static void MultiplyArrayByVectorImpl( MultiTensor &aOut, MultiTensor &aIn, MemoryBuffer &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::MultiplyArrayByVector<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    void MultiplyOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, MultiplyArrayByScalarImpl, ( aOut, aLeft, aRight ) );
    }

    void MultiplyOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, MultiplyArrayByArrayImpl, ( aOut, aLeft, aRight ) );
    }

    void MultiplyOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight,
                     eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                     uint32_t aMaxBroadcastSizes )
    {
        DISPATCH_BY_TYPE( aTensorElementType, MultiplyArrayByArrayImpl,
                          ( aOut, aLeft, aRight, aBroadcastHint, aBlockSizes, aMaxBlockSize, aBroadcastSizes, aMaxBroadcastSizes ) );
    }

    void MultiplyOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, MultiplyArrayByVectorImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    void SubtractArrayFromScalarImpl( MultiTensor &aOut, scalar_value_t &aConstant, MultiTensor &aIn )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::SubtractArrayFromScalar<<<lGridDim, lBlockDim>>>( aOut, aIn, std::get<_Ty>( aConstant ) );
    }

    template <typename _Ty>
    void SubtractScalarFromArrayImpl( MultiTensor &aOut, MultiTensor &aIn, scalar_value_t &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::SubtractScalarFromArray<<<lGridDim, lBlockDim>>>( aOut, aIn, std::get<_Ty>( aConstant ) );
    }

    template <typename _Ty>
    static void SubtractVectorFromArrayImpl( MultiTensor &aOut, MultiTensor &aIn, MemoryBuffer &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::SubtractVectorFromArray<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    template <typename _Ty>
    static void SubtractArrayfromArrayImpl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::SubtractArrayFromArray<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    template <typename _Ty>
    static void SubtractArrayfromArrayImpl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant, eBroadcastHint aBroadcastHint,
                                            MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                                            uint32_t aMaxBroadcastSizes )
    {
        int lBlockCount = ( aMaxBroadcastSizes / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), aMaxBlockSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::SubtractArrayFromArray<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant, aBroadcastHint, aBlockSizes, aBroadcastSizes );
    }

    template <typename _Ty>
    static void SubtractArrayFromVectorImpl( MultiTensor &aOut, MemoryBuffer &aConstant, MultiTensor &aIn )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::SubtractArrayFromVector<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    void SubtractOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, SubtractScalarFromArrayImpl, ( aOut, aLeft, aRight ) );
    }

    void SubtractOp( scalar_type_t aTensorElementType, MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, SubtractArrayFromScalarImpl, ( aOut, aLeft, aRight ) );
    }

    void SubtractOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, SubtractArrayfromArrayImpl, ( aOut, aLeft, aRight ) );
    }

    void SubtractOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight,
                     eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                     uint32_t aMaxBroadcastSizes )
    {
        DISPATCH_BY_TYPE( aTensorElementType, SubtractArrayfromArrayImpl,
                          ( aOut, aLeft, aRight, aBroadcastHint, aBlockSizes, aMaxBlockSize, aBroadcastSizes, aMaxBroadcastSizes ) );
    }

    void SubtractOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, SubtractVectorFromArrayImpl, ( aOut, aLeft, aRight ) );
    }

    void SubtractOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, SubtractArrayFromVectorImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void DivideArrayByScalarImpl( MultiTensor &aOut, MultiTensor &aIn, scalar_value_t &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::DivideArrayByScalar<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, std::get<_Ty>( aConstant ) );
    }

    template <typename _Ty>
    static void DivideScalarByArrayImpl( MultiTensor &aOut, scalar_value_t &aConstant, MultiTensor &aIn )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::DivideScalarByArray<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, std::get<_Ty>( aConstant ) );
    }

    template <typename _Ty>
    static void DivideArrayfromArrayImpl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::DivideArrayFromArray<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    template <typename _Ty>
    static void DivideArrayfromArrayImpl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant, eBroadcastHint aBroadcastHint,
                                          MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                                          uint32_t aMaxBroadcastSizes )
    {
        int lBlockCount = ( aMaxBroadcastSizes / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), aMaxBlockSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::DivideArrayFromArray<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant, aBroadcastHint, aBlockSizes, aBroadcastSizes );
    }

    template <typename _Ty>
    static void DivideArrayByVectorImpl( MultiTensor &aOut, MultiTensor &aIn, MemoryBuffer &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::DivideArrayByVector<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    template <typename _Ty>
    static void DivideVectorByArrayImpl( MultiTensor &aOut, MemoryBuffer &aConstant, MultiTensor &aIn )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::DivideVectorByArray<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    void DivideOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, DivideArrayByScalarImpl, ( aOut, aLeft, aRight ) );
    }

    void DivideOp( scalar_type_t aTensorElementType, MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, DivideScalarByArrayImpl, ( aOut, aLeft, aRight ) );
    }

    void DivideOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, DivideArrayfromArrayImpl, ( aOut, aLeft, aRight ) );
    }

    void DivideOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight,
                   eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                   uint32_t aMaxBroadcastSizes )
    {
        DISPATCH_BY_TYPE( aTensorElementType, DivideArrayfromArrayImpl,
                          ( aOut, aLeft, aRight, aBroadcastHint, aBlockSizes, aMaxBlockSize, aBroadcastSizes, aMaxBroadcastSizes ) );
    }

    void DivideOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, DivideArrayByVectorImpl, ( aOut, aLeft, aRight ) );
    }

    void DivideOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, DivideVectorByArrayImpl, ( aOut, aLeft, aRight ) );
    }

    void AndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AndTensorScalar<<<lGridDim, lBlockDim>>>( aOut, aLeft, std::get<uint8_t>( aRight ) );
    }

    void AndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        AndOp( aTensorElementType, aOut, aRight, aLeft );
    }

    void AndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant,
                eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                uint32_t aMaxBroadcastSizes )
    {
        int lBlockCount = ( aMaxBroadcastSizes / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), aMaxBlockSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AndTensorTensor<<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant, aBroadcastHint, aBlockSizes, aBroadcastSizes );
    }

    void AndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AndTensorTensor<<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void AndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AndTensorVector<<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void AndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        AndOp( aTensorElementType, aOut, aRight, aLeft );
    }

    void OrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::OrTensorScalar<<<lGridDim, lBlockDim>>>( aOut, aLeft, std::get<uint8_t>( aRight ) );
    }

    void OrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        OrOp( aTensorElementType, aOut, aRight, aLeft );
    }

    void OrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::OrTensorTensor<<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void OrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant,
               eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
               uint32_t aMaxBroadcastSizes )
    {
        int lBlockCount = ( aMaxBroadcastSizes / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), aMaxBlockSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::OrTensorTensor<<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant, aBroadcastHint, aBlockSizes, aBroadcastSizes );
    }

    void OrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::OrTensorVector<<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void OrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        OrOp( aTensorElementType, aOut, aRight, aLeft );
    }

    void NotOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aOperand )
    {
        int lBlockCount = ( aOperand.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aOperand.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::NotTensor<<<lGridDim, lBlockDim>>>( aOut, aOperand );
    }

    template <typename _Ty>
    void BitwiseAnd_Tensor_Scalar_Impl( MultiTensor &aOut, MultiTensor &aIn, scalar_value_t &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::BitwiseAndTensorScalar<<<lGridDim, lBlockDim>>>( aOut, aIn, std::get<_Ty>( aConstant ) );
    }

    void BitwiseAndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        DISPATCH_BY_INTEGRAL_TYPE( aTensorElementType, BitwiseAnd_Tensor_Scalar_Impl, ( aOut, aLeft, aRight ) );
    }

    void BitwiseAndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        BitwiseAndOp( aTensorElementType, aOut, aRight, aLeft );
    }

    template <typename _Ty>
    static void BitwiseAnd_Tensor_Tensor_Impl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant,
                                               eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize,
                                               MemoryBuffer &aBroadcastSizes, uint32_t aMaxBroadcastSizes )
    {
        int lBlockCount = ( aMaxBroadcastSizes / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), aMaxBlockSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::BitwiseAndTensorTensor<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant, aBroadcastHint, aBlockSizes, aBroadcastSizes );
    }

    void BitwiseAndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight,
                       eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                       uint32_t aMaxBroadcastSizes )
    {
        DISPATCH_BY_INTEGRAL_TYPE(
            aTensorElementType, BitwiseAnd_Tensor_Tensor_Impl,
            ( aOut, aLeft, aRight, aBroadcastHint, aBlockSizes, aMaxBlockSize, aBroadcastSizes, aMaxBroadcastSizes ) );
    }

    template <typename _Ty>
    void BitwiseAnd_Tensor_Tensor_Impl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::BitwiseAndTensorTensor<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    void BitwiseAndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_INTEGRAL_TYPE( aTensorElementType, BitwiseAnd_Tensor_Tensor_Impl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    void BitwiseAnd_Tensor_Vector_Impl( MultiTensor &aOut, MultiTensor &aIn, MemoryBuffer &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::BitwiseAnd_Tensor_Vector<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    void BitwiseAndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        DISPATCH_BY_INTEGRAL_TYPE( aTensorElementType, BitwiseAnd_Tensor_Vector_Impl, ( aOut, aLeft, aRight ) );
    }

    void BitwiseAndOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        BitwiseAndOp( aTensorElementType, aOut, aRight, aLeft );
    }

    template <typename _Ty>
    void BitwiseOr_Tensor_Scalar_Impl( MultiTensor &aOut, MultiTensor &aIn, scalar_value_t &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::BitwiseOrTensorScalar<<<lGridDim, lBlockDim>>>( aOut, aIn, std::get<_Ty>( aConstant ) );
    }

    void BitwiseOrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        DISPATCH_BY_INTEGRAL_TYPE( aTensorElementType, BitwiseOr_Tensor_Scalar_Impl, ( aOut, aLeft, aRight ) );
    }

    void BitwiseOrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        BitwiseOrOp( aTensorElementType, aOut, aRight, aLeft );
    }

    template <typename _Ty>
    void BitwiseOr_Tensor_Tensor_Impl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::BitwiseOrTensorTensor<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    void BitwiseOrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_INTEGRAL_TYPE( aTensorElementType, BitwiseOr_Tensor_Tensor_Impl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void BitwiseOr_Tensor_Tensor_Impl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant,
                                              eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize,
                                              MemoryBuffer &aBroadcastSizes, uint32_t aMaxBroadcastSizes )
    {
        int lBlockCount = ( aMaxBroadcastSizes / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), aMaxBlockSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::BitwiseOrTensorTensor<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant, aBroadcastHint, aBlockSizes, aBroadcastSizes );
    }

    void BitwiseOrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight,
                      eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                      uint32_t aMaxBroadcastSizes )
    {
        DISPATCH_BY_INTEGRAL_TYPE(
            aTensorElementType, BitwiseOr_Tensor_Tensor_Impl,
            ( aOut, aLeft, aRight, aBroadcastHint, aBlockSizes, aMaxBlockSize, aBroadcastSizes, aMaxBroadcastSizes ) );
    }

    template <typename _Ty>
    void BitwiseOrTensorVectorImpl( MultiTensor &aOut, MultiTensor &aIn, MemoryBuffer &aConstant )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::BitwiseOrTensorVector<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant );
    }

    void BitwiseOrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        DISPATCH_BY_INTEGRAL_TYPE( aTensorElementType, BitwiseOrTensorVectorImpl, ( aOut, aLeft, aRight ) );
    }

    void BitwiseOrOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        BitwiseOrOp( aTensorElementType, aOut, aRight, aLeft );
    }

    template <typename _Ty>
    void BitwiseNotTensorImpl( MultiTensor &aOut, MultiTensor &aIn )
    {
        int lBlockCount = ( aIn.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::BitwiseNotTensor<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn );
    }

    void BitwiseNotOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aOperand )
    {
        DISPATCH_BY_INTEGRAL_TYPE( aTensorElementType, BitwiseNotTensorImpl, ( aOut, aOperand ) );
    }

    template <typename _Ty>
    static void EqualOpImpl( MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::EqualOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void EqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, EqualOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void EqualOpImpl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant, eBroadcastHint aBroadcastHint,
                             MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                             uint32_t aMaxBroadcastSizes )
    {
        int lBlockCount = ( aMaxBroadcastSizes / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), aMaxBlockSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::EqualOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant, aBroadcastHint, aBlockSizes, aBroadcastSizes );
    }

    void EqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight,
                  eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                  uint32_t aMaxBroadcastSizes )
    {
        DISPATCH_BY_TYPE( aTensorElementType, EqualOpImpl,
                          ( aOut, aLeft, aRight, aBroadcastHint, aBlockSizes, aMaxBlockSize, aBroadcastSizes, aMaxBroadcastSizes ) );
    }

    template <typename _Ty>
    static void EqualOpImpl( MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::EqualOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void EqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, EqualOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _ScalarType>
    static void EqualOpImpl( MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::EqualOp<_ScalarType><<<lGridDim, lBlockDim>>>( aOut, aLeft, std::get<_ScalarType>( aRight ) );
    }

    void EqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, EqualOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void EqualOpImpl( MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        int lBlockCount = ( aRight.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aRight.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::EqualOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void EqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, EqualOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void EqualOpImpl( MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        int lBlockCount = ( aRight.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aRight.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::EqualOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, std::get<_Ty>( aLeft ), aRight );
    }

    void EqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, EqualOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void LessThanOpImpl( MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void LessThanOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void LessThanOpImpl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant, eBroadcastHint aBroadcastHint,
                                MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                                uint32_t aMaxBroadcastSizes )
    {
        int lBlockCount = ( aMaxBroadcastSizes / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), aMaxBlockSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant, aBroadcastHint, aBlockSizes, aBroadcastSizes );
    }

    void LessThanOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight,
                     eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                     uint32_t aMaxBroadcastSizes )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOpImpl,
                          ( aOut, aLeft, aRight, aBroadcastHint, aBlockSizes, aMaxBlockSize, aBroadcastSizes, aMaxBroadcastSizes ) );
    }

    template <typename _Ty>
    static void LessThanOpImpl( MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void LessThanOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _ScalarType>
    static void LessThanOpImpl( MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOp<_ScalarType><<<lGridDim, lBlockDim>>>( aOut, aLeft, std::get<_ScalarType>( aRight ) );
    }

    void LessThanOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void LessThanOpImpl( MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        int lBlockCount = ( aRight.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aRight.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void LessThanOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void LessThanOpImpl( MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        int lBlockCount = ( aRight.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aRight.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, std::get<_Ty>( aLeft ), aRight );
    }

    void LessThanOp( scalar_type_t aTensorElementType, MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void LessThanOrEqualOpImpl( MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOrEqualOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void LessThanOrEqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOrEqualOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void LessThanOrEqualOpImpl( MultiTensor &aOut, MultiTensor &aIn, MultiTensor &aConstant, eBroadcastHint aBroadcastHint,
                                       MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize, MemoryBuffer &aBroadcastSizes,
                                       uint32_t aMaxBroadcastSizes )
    {
        int lBlockCount = ( aMaxBroadcastSizes / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aIn.Shape().CountLayers(), aMaxBlockSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOrEqualOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aIn, aConstant, aBroadcastHint, aBlockSizes, aBroadcastSizes );
    }

    void LessThanOrEqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight,
                            eBroadcastHint aBroadcastHint, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize,
                            MemoryBuffer &aBroadcastSizes, uint32_t aMaxBroadcastSizes )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOrEqualOpImpl,
                          ( aOut, aLeft, aRight, aBroadcastHint, aBlockSizes, aMaxBlockSize, aBroadcastSizes, aMaxBroadcastSizes ) );
    }

    template <typename _Ty>
    static void LessThanOrEqualOpImpl( MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOrEqualOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void LessThanOrEqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MemoryBuffer &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOrEqualOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _ScalarType>
    static void LessThanOrEqualOpImpl( MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        int lBlockCount = ( aLeft.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOrEqualOp<_ScalarType><<<lGridDim, lBlockDim>>>( aOut, aLeft, std::get<_ScalarType>( aRight ) );
    }

    void LessThanOrEqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, scalar_value_t &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOrEqualOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void LessThanOrEqualOpImpl( MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        int lBlockCount = ( aRight.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aRight.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOrEqualOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight );
    }

    void LessThanOrEqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MemoryBuffer &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOrEqualOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void LessThanOrEqualOpImpl( MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        int lBlockCount = ( aRight.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aRight.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LessThanOrEqualOp<_Ty><<<lGridDim, lBlockDim>>>( aOut, std::get<_Ty>( aLeft ), aRight );
    }

    void LessThanOrEqualOp( scalar_type_t aTensorElementType, MultiTensor &aOut, scalar_value_t &aLeft, MultiTensor &aRight )
    {
        DISPATCH_BY_TYPE( aTensorElementType, LessThanOrEqualOpImpl, ( aOut, aLeft, aRight ) );
    }

    template <typename _Ty>
    static void InIntervalTensorTensorImpl( MultiTensor &aOut, MultiTensor &aX, MultiTensor &aLower, MultiTensor &aUpper,
                                            bool aStrictLower, bool aStrictUpper )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::InIntervalTensorTensor<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper );
    }

    void InIntervalOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aX, MultiTensor &aLower, MultiTensor &aUpper,
                       bool aStrictLower, bool aStrictUpper )
    {
        DISPATCH_BY_TYPE( aTensorElementType, InIntervalTensorTensorImpl, ( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper ) );
    }

    template <typename _Ty>
    static void InIntervalTensorVectorImpl( MultiTensor &aOut, MultiTensor &aX, MultiTensor &aLower, MemoryBuffer &aUpper,
                                            bool aStrictLower, bool aStrictUpper )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::InIntervalTensorVector<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper );
    }

    void InIntervalOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aX, MultiTensor &aLower, MemoryBuffer &aUpper,
                       bool aStrictLower, bool aStrictUpper )
    {
        DISPATCH_BY_TYPE( aTensorElementType, InIntervalTensorVectorImpl, ( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper ) );
    }

    template <typename _Ty>
    static void InIntervalTensorScalarImpl( MultiTensor &aOut, MultiTensor &aX, MultiTensor &aLower, scalar_value_t &aUpper,
                                            bool aStrictLower, bool aStrictUpper )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::InIntervalTensorScalar<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aX, aLower, std::get<_Ty>( aUpper ), aStrictLower, aStrictUpper );
    }

    void InIntervalOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aX, MultiTensor &aLower, scalar_value_t &aUpper,
                       bool aStrictLower, bool aStrictUpper )
    {
        DISPATCH_BY_TYPE( aTensorElementType, InIntervalTensorScalarImpl, ( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper ) );
    }

    template <typename _Ty>
    static void InIntervalVectorTensorImpl( MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aLower, MultiTensor &aUpper,
                                            bool aStrictLower, bool aStrictUpper )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::InIntervalVectorTensor<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper );
    }

    void InIntervalOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aLower, MultiTensor &aUpper,
                       bool aStrictLower, bool aStrictUpper )
    {
        DISPATCH_BY_TYPE( aTensorElementType, InIntervalVectorTensorImpl, ( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper ) );
    }

    template <typename _Ty>
    static void InIntervalVectorVectorImpl( MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aLower, MemoryBuffer &aUpper,
                                            bool aStrictLower, bool aStrictUpper )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::InIntervalVectorVector<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper );
    }

    void InIntervalOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aLower, MemoryBuffer &aUpper,
                       bool aStrictLower, bool aStrictUpper )
    {
        DISPATCH_BY_TYPE( aTensorElementType, InIntervalVectorVectorImpl, ( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper ) );
    }

    template <typename _Ty>
    static void InIntervalVectorScalarImpl( MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aLower, scalar_value_t &aUpper,
                                            bool aStrictLower, bool aStrictUpper )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::InIntervalVectorScalar<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aX, aLower, std::get<_Ty>( aUpper ), aStrictLower, aStrictUpper );
    }

    void InIntervalOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aLower, scalar_value_t &aUpper,
                       bool aStrictLower, bool aStrictUpper )
    {
        DISPATCH_BY_TYPE( aTensorElementType, InIntervalVectorScalarImpl, ( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper ) );
    }

    template <typename _Ty>
    static void InIntervalScalarTensorImpl( MultiTensor &aOut, MultiTensor &aX, scalar_value_t &aLower, MultiTensor &aUpper,
                                            bool aStrictLower, bool aStrictUpper )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::InIntervalScalarTensor<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aX, std::get<_Ty>( aLower ), aUpper, aStrictLower, aStrictUpper );
    }

    void InIntervalOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aX, scalar_value_t &aLower, MultiTensor &aUpper,
                       bool aStrictLower, bool aStrictUpper )
    {
        DISPATCH_BY_TYPE( aTensorElementType, InIntervalScalarTensorImpl, ( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper ) );
    }

    template <typename _Ty>
    static void InIntervalScalarVectorImpl( MultiTensor &aOut, MultiTensor &aX, scalar_value_t &aLower, MemoryBuffer &aUpper,
                                            bool aStrictLower, bool aStrictUpper )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::InIntervalScalarVector<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aX, std::get<_Ty>( aLower ), aUpper, aStrictLower, aStrictUpper );
    }

    void InIntervalOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aX, scalar_value_t &aLower, MemoryBuffer &aUpper,
                       bool aStrictLower, bool aStrictUpper )
    {
        DISPATCH_BY_TYPE( aTensorElementType, InIntervalScalarVectorImpl, ( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper ) );
    }

    template <typename _Ty>
    static void InIntervalScalarScalarImpl( MultiTensor &aOut, MultiTensor &aX, scalar_value_t &aLower, scalar_value_t &aUpper,
                                            bool aStrictLower, bool aStrictUpper )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::InIntervalScalarScalar<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aX, std::get<_Ty>( aLower ), std::get<_Ty>( aUpper ), aStrictLower, aStrictUpper );
    }

    void InIntervalOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aX, scalar_value_t &aLower, scalar_value_t &aUpper,
                       bool aStrictLower, bool aStrictUpper )
    {
        DISPATCH_BY_TYPE( aTensorElementType, InIntervalScalarScalarImpl, ( aOut, aX, aLower, aUpper, aStrictLower, aStrictUpper ) );
    }

    template <typename _Ty>
    static void WhereOpTensorTensorImpl( MultiTensor &aOut, MultiTensor &aCondition, MultiTensor &aValueIfTrue,
                                         MultiTensor &aValueIfFalse )
    {
        int lBlockCount = ( aCondition.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aCondition.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::WhereTensorTensor<_Ty><<<lGridDim, lBlockDim>>>( aOut, aCondition, aValueIfTrue, aValueIfFalse );
    }

    void WhereOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aCondition, MultiTensor &aValueIfTrue,
                  MultiTensor &aValueIfFalse )
    {
        DISPATCH_BY_TYPE( aTensorElementType, WhereOpTensorTensorImpl, ( aOut, aCondition, aValueIfTrue, aValueIfFalse ) );
    }

    template <typename _Ty>
    static void WhereTensorVectorImpl( MultiTensor &aOut, MultiTensor &aCondition, MultiTensor &aValueIfTrue,
                                       MemoryBuffer &aValueIfFalse )
    {
        int lBlockCount = ( aCondition.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aCondition.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::WhereTensorVector<_Ty><<<lGridDim, lBlockDim>>>( aOut, aCondition, aValueIfTrue, aValueIfFalse );
    }

    void WhereOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aCondition, MultiTensor &aValueIfTrue,
                  MemoryBuffer &aValueIfFalse )
    {
        DISPATCH_BY_TYPE( aTensorElementType, WhereTensorVectorImpl, ( aOut, aCondition, aValueIfTrue, aValueIfFalse ) );
    }

    template <typename _Ty>
    static void WhereTensorScalarImpl( MultiTensor &aOut, MultiTensor &aCondition, MultiTensor &aValueIfTrue,
                                       scalar_value_t &aValueIfFalse )
    {
        int lBlockCount = ( aCondition.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aCondition.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::WhereTensorScalar<_Ty><<<lGridDim, lBlockDim>>>( aOut, aCondition, aValueIfTrue, std::get<_Ty>( aValueIfFalse ) );
    }

    void WhereOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aCondition, MultiTensor &aValueIfTrue,
                  scalar_value_t &aValueIfFalse )
    {
        DISPATCH_BY_TYPE( aTensorElementType, WhereTensorScalarImpl, ( aOut, aCondition, aValueIfTrue, aValueIfFalse ) );
    }

    template <typename _Ty>
    static void WhereVectorTensorImpl( MultiTensor &aOut, MultiTensor &aCondition, MemoryBuffer &aValueIfTrue,
                                       MultiTensor &aValueIfFalse )
    {
        int lBlockCount = ( aCondition.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aCondition.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::WhereVectorTensor<_Ty><<<lGridDim, lBlockDim>>>( aOut, aCondition, aValueIfTrue, aValueIfFalse );
    }
    void WhereOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aCondition, MemoryBuffer &aValueIfTrue,
                  MultiTensor &aValueIfFalse )
    {
        DISPATCH_BY_TYPE( aTensorElementType, WhereVectorTensorImpl, ( aOut, aCondition, aValueIfTrue, aValueIfFalse ) );
    }

    template <typename _Ty>
    static void WhereVectorVectorImpl( MultiTensor &aOut, MultiTensor &aCondition, MemoryBuffer &aValueIfTrue,
                                       MemoryBuffer &aValueIfFalse )
    {
        int lBlockCount = ( aCondition.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aCondition.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::WhereVectorVector<_Ty><<<lGridDim, lBlockDim>>>( aOut, aCondition, aValueIfTrue, aValueIfFalse );
    }

    void WhereOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aCondition, MemoryBuffer &aValueIfTrue,
                  MemoryBuffer &aValueIfFalse )
    {
        DISPATCH_BY_TYPE( aTensorElementType, WhereVectorVectorImpl, ( aOut, aCondition, aValueIfTrue, aValueIfFalse ) );
    }

    template <typename _Ty>
    static void WhereVectorScalarImpl( MultiTensor &aOut, MultiTensor &aCondition, MemoryBuffer &aValueIfTrue,
                                       scalar_value_t &aValueIfFalse )
    {
        int lBlockCount = ( aCondition.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aCondition.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::WhereVectorScalar<_Ty><<<lGridDim, lBlockDim>>>( aOut, aCondition, aValueIfTrue, std::get<_Ty>( aValueIfFalse ) );
    }

    void WhereOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aCondition, MemoryBuffer &aValueIfTrue,
                  scalar_value_t &aValueIfFalse )
    {
        DISPATCH_BY_TYPE( aTensorElementType, WhereVectorScalarImpl, ( aOut, aCondition, aValueIfTrue, aValueIfFalse ) );
    }

    template <typename _Ty>
    static void WhereScalarTensorImpl( MultiTensor &aOut, MultiTensor &aCondition, scalar_value_t &aValueIfTrue,
                                       MultiTensor &aValueIfFalse )
    {
        int lBlockCount = ( aCondition.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aCondition.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::WhereScalarTensor<_Ty><<<lGridDim, lBlockDim>>>( aOut, aCondition, std::get<_Ty>( aValueIfTrue ), aValueIfFalse );
    }

    void WhereOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aCondition, scalar_value_t &aValueIfTrue,
                  MultiTensor &aValueIfFalse )
    {
        DISPATCH_BY_TYPE( aTensorElementType, WhereScalarTensorImpl, ( aOut, aCondition, aValueIfTrue, aValueIfFalse ) );
    }

    template <typename _Ty>
    static void WhereScalarVectorImpl( MultiTensor &aOut, MultiTensor &aCondition, scalar_value_t &aValueIfTrue,
                                       MemoryBuffer &aValueIfFalse )
    {
        int lBlockCount = ( aCondition.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aCondition.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::WhereScalarVector<_Ty><<<lGridDim, lBlockDim>>>( aOut, aCondition, std::get<_Ty>( aValueIfTrue ), aValueIfFalse );
    }

    void WhereOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aCondition, scalar_value_t &aValueIfTrue,
                  MemoryBuffer &aValueIfFalse )
    {
        DISPATCH_BY_TYPE( aTensorElementType, WhereScalarVectorImpl, ( aOut, aCondition, aValueIfTrue, aValueIfFalse ) );
    }

    template <typename _Ty>
    static void WhereScalarScalarImpl( MultiTensor &aOut, MultiTensor &aCondition, scalar_value_t &aValueIfTrue,
                                       scalar_value_t &aValueIfFalse )
    {
        int lBlockCount = ( aCondition.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aCondition.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::WhereScalarScalar<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aCondition, std::get<_Ty>( aValueIfTrue ), std::get<_Ty>( aValueIfFalse ) );
    }

    void WhereOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aCondition, scalar_value_t &aValueIfTrue,
                  scalar_value_t &aValueIfFalse )
    {
        DISPATCH_BY_TYPE( aTensorElementType, WhereScalarScalarImpl, ( aOut, aCondition, aValueIfTrue, aValueIfFalse ) );
    }

    template <typename _Ty>
    static void RepeatOpImpl( MultiTensor &aOut, MultiTensor &aArray, MemoryBuffer &aRepetitions, uint32_t lMaxRepetitions )
    {
        int lBlockCount = ( lMaxRepetitions / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aArray.Shape().CountLayers(), aArray.Shape().mMaxBufferSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Repeat<_Ty><<<lGridDim, lBlockDim>>>( aOut, aArray, aRepetitions );
    }

    void RepeatOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aArray, MemoryBuffer &aRepetitions,
                   uint32_t lMaxRepetitions )
    {
        DISPATCH_BY_TYPE( aTensorElementType, RepeatOpImpl, ( aOut, aArray, aRepetitions, lMaxRepetitions ) );
    }

    template <typename _Ty>
    static void TileOpImpl( MultiTensor &aOut, MultiTensor &aArray, MemoryBuffer &aRepetitions, uint32_t lMaxRepetitions )
    {
        int lBlockCount = ( aArray.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aArray.Shape().CountLayers(), lMaxRepetitions, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Tile<_Ty><<<lGridDim, lBlockDim>>>( aOut, aArray, aRepetitions );
    }

    void TileOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aArray, MemoryBuffer &aRepetitions,
                 uint32_t lMaxRepetitions )
    {
        DISPATCH_BY_TYPE( aTensorElementType, TileOpImpl, ( aOut, aArray, aRepetitions, lMaxRepetitions ) );
    }

    template <typename _Ty>
    static void LinearSpaceOpImpl( MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight, MemoryBuffer &aSubdivisions,
                                   uint32_t aMaxSubdivisions )
    {
        int lBlockCount = ( aMaxSubdivisions / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aLeft.Shape().CountLayers(), aLeft.Shape().mMaxBufferSize, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::LinearSpace<_Ty><<<lGridDim, lBlockDim>>>( aOut, aLeft, aRight, aSubdivisions );
    }

    void LinearSpaceOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &aLeft, MultiTensor &aRight,
                        MemoryBuffer &aSubdivisions, uint32_t aMaxSubdivisions )
    {
        switch( aTensorElementType )
        {
        case scalar_type_t::FLOAT32:
        {
            LinearSpaceOpImpl<float>( aOut, aLeft, aRight, aSubdivisions, aMaxSubdivisions );
            break;
        }
        case scalar_type_t::FLOAT64:
        {
            LinearSpaceOpImpl<double>( aOut, aLeft, aRight, aSubdivisions, aMaxSubdivisions );
            break;
        }
        default:
            throw std::runtime_error( "Linear space only supports float and double values" );
        }
    }

    template <typename _Ty>
    static void MixImpl( MultiTensor &aOut, MultiTensor &A, MultiTensor &B, MultiTensor &t )
    {
        int lBlockCount = ( A.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( A.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Mix<_Ty><<<lGridDim, lBlockDim>>>( aOut, A, B, t );
    }

    void MixOp( scalar_type_t aTensorElementType, MultiTensor &aOut, MultiTensor &A, MultiTensor &B, MultiTensor &t )
    {
        DISPATCH_BY_TYPE( aTensorElementType, MixImpl, ( aOut, A, B, t ) );
    }

    void Sample2DOp( MultiTensor &aOut, MultiTensor &X, MultiTensor &Y, MemoryBuffer &aTextures )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Sample2D<<<lGridDim, lBlockDim>>>( aOut, X, Y, aTextures );
    }

    void Sample2DOp( MultiTensor &aOut, MultiTensor &X, MemoryBuffer &Y, MemoryBuffer &aTextures )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Sample2D<<<lGridDim, lBlockDim>>>( aOut, X, Y, aTextures );
    }

    void Sample2DOp( MultiTensor &aOut, MultiTensor &X, scalar_value_t &Y, MemoryBuffer &aTextures )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Sample2D<<<lGridDim, lBlockDim>>>( aOut, X, std::get<float>( Y ), aTextures );
    }

    void Sample2DOp( MultiTensor &aOut, MemoryBuffer &X, MultiTensor &Y, MemoryBuffer &aTextures )
    {
        int lBlockCount = ( Y.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( Y.Shape().CountLayers(), lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Sample2D<<<lGridDim, lBlockDim>>>( aOut, X, Y, aTextures );
    }

    void Sample2DOp( MultiTensor &aOut, scalar_value_t &X, MultiTensor &Y, MemoryBuffer &aTextures )
    {
        int lBlockCount = ( Y.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( Y.Shape().CountLayers(), lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Sample2D<<<lGridDim, lBlockDim>>>( aOut, std::get<float>( X ), Y, aTextures );
    }

    template <typename _Ty>
    static void ToFixedPointOpImpl( MultiTensor &aOut, scalar_type_t aOutputElementType, MultiTensor &aArray, _Ty aScaling )
    {
        int lBlockCount = ( aArray.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aArray.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        switch( aOutputElementType )
        {
        case scalar_type_t::UINT8:
        {
            Kernels::ToFixedPoint<_Ty, uint8_t><<<lGridDim, lBlockDim>>>( aOut, aArray, aScaling );
            break;
        }
        case scalar_type_t::UINT16:
        {
            Kernels::ToFixedPoint<_Ty, uint16_t><<<lGridDim, lBlockDim>>>( aOut, aArray, aScaling );
            break;
        }
        case scalar_type_t::UINT32:
        {
            Kernels::ToFixedPoint<_Ty, uint32_t><<<lGridDim, lBlockDim>>>( aOut, aArray, aScaling );
            break;
        }
        case scalar_type_t::UINT64:
        {
            Kernels::ToFixedPoint<_Ty, uint64_t><<<lGridDim, lBlockDim>>>( aOut, aArray, aScaling );
            break;
        }
        case scalar_type_t::INT8:
        {
            Kernels::ToFixedPoint<_Ty, int8_t><<<lGridDim, lBlockDim>>>( aOut, aArray, aScaling );
            break;
        }
        case scalar_type_t::INT16:
        {
            Kernels::ToFixedPoint<_Ty, int16_t><<<lGridDim, lBlockDim>>>( aOut, aArray, aScaling );
            break;
        }
        case scalar_type_t::INT32:
        {
            Kernels::ToFixedPoint<_Ty, int32_t><<<lGridDim, lBlockDim>>>( aOut, aArray, aScaling );
            break;
        }
        case scalar_type_t::INT64:
        {
            Kernels::ToFixedPoint<_Ty, int64_t><<<lGridDim, lBlockDim>>>( aOut, aArray, aScaling );
            break;
        }
        default:
            throw std::runtime_error( "Linear space only supports float and double values" );
        }
    }

    void ToFixedPointOp( scalar_type_t aTensorElementType, MultiTensor &aOut, scalar_type_t aOutputElementType, MultiTensor &aArray,
                         scalar_value_t &aScaling )
    {
        switch( aTensorElementType )
        {
        case scalar_type_t::FLOAT32:
        {
            ToFixedPointOpImpl<float>( aOut, aOutputElementType, aArray, std::get<float>( aScaling ) );
            break;
        }
        case scalar_type_t::FLOAT64:
        {
            ToFixedPointOpImpl<double>( aOut, aOutputElementType, aArray, std::get<double>( aScaling ) );
            break;
        }
        default:
            throw std::runtime_error( "Linear space only supports float and double values" );
        }
    }

    template <typename _Ty>
    static void AffineTransformImpl( MultiTensor &aOut, MultiTensor &A, MultiTensor &X, MultiTensor &B )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AffineTransform<_Ty><<<lGridDim, lBlockDim>>>( aOut, A, X, B );
    }

    template <typename _Ty>
    static void AffineTransformImpl( MultiTensor &aOut, MultiTensor &A, MultiTensor &X, MemoryBuffer &B )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AffineTransform<_Ty><<<lGridDim, lBlockDim>>>( aOut, A, X, B );
    }

    template <typename _Ty>
    static void AffineTransformImpl( MultiTensor &aOut, MultiTensor &A, MultiTensor &X, scalar_value_t &B )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AffineTransform<_Ty><<<lGridDim, lBlockDim>>>( aOut, A, X, std::get<_Ty>( B ) );
    }

    template <typename _Ty>
    static void AffineTransformImpl( MultiTensor &aOut, MemoryBuffer &A, MultiTensor &X, MultiTensor &B )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AffineTransform<_Ty><<<lGridDim, lBlockDim>>>( aOut, A, X, B );
    }

    template <typename _Ty>
    static void AffineTransformImpl( MultiTensor &aOut, MemoryBuffer &A, MultiTensor &X, MemoryBuffer &B )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AffineTransform<_Ty><<<lGridDim, lBlockDim>>>( aOut, A, X, B );
    }

    template <typename _Ty>
    static void AffineTransformImpl( MultiTensor &aOut, MemoryBuffer &A, MultiTensor &X, scalar_value_t &B )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AffineTransform<_Ty><<<lGridDim, lBlockDim>>>( aOut, A, X, std::get<_Ty>( B ) );
    }

    template <typename _Ty>
    static void AffineTransformImpl( MultiTensor &aOut, scalar_value_t &A, MultiTensor &X, MultiTensor &B )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AffineTransform<_Ty><<<lGridDim, lBlockDim>>>( aOut, std::get<_Ty>( A ), X, B );
    }

    template <typename _Ty>
    static void AffineTransformImpl( MultiTensor &aOut, scalar_value_t &A, MultiTensor &X, MemoryBuffer &B )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AffineTransform<_Ty><<<lGridDim, lBlockDim>>>( aOut, std::get<_Ty>( A ), X, B );
    }

    template <typename _Ty>
    static void AffineTransformImpl( MultiTensor &aOut, scalar_value_t &A, MultiTensor &X, scalar_value_t &B )
    {
        int lBlockCount = ( X.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( X.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::AffineTransform<_Ty><<<lGridDim, lBlockDim>>>( aOut, std::get<_Ty>( A ), X, std::get<_Ty>( B ) );
    }

    void AffineTransformOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &A, MultiTensor &X, MultiTensor &B )
    {
        DISPATCH_BY_TYPE( aOutputElementType, AffineTransformImpl, ( aOut, A, X, B ) );
    }

    void AffineTransformOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &A, MultiTensor &X, MemoryBuffer &B )
    {
        DISPATCH_BY_TYPE( aOutputElementType, AffineTransformImpl, ( aOut, A, X, B ) );
    }

    void AffineTransformOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &A, MultiTensor &X, scalar_value_t &B )
    {
        DISPATCH_BY_TYPE( aOutputElementType, AffineTransformImpl, ( aOut, A, X, B ) );
    }

    void AffineTransformOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MemoryBuffer &A, MultiTensor &X, MultiTensor &B )
    {
        DISPATCH_BY_TYPE( aOutputElementType, AffineTransformImpl, ( aOut, A, X, B ) );
    }

    void AffineTransformOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MemoryBuffer &A, MultiTensor &X, MemoryBuffer &B )
    {
        DISPATCH_BY_TYPE( aOutputElementType, AffineTransformImpl, ( aOut, A, X, B ) );
    }

    void AffineTransformOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MemoryBuffer &A, MultiTensor &X, scalar_value_t &B )
    {
        DISPATCH_BY_TYPE( aOutputElementType, AffineTransformImpl, ( aOut, A, X, B ) );
    }

    void AffineTransformOp( scalar_type_t aOutputElementType, MultiTensor &aOut, scalar_value_t &A, MultiTensor &X, MultiTensor &B )
    {
        DISPATCH_BY_TYPE( aOutputElementType, AffineTransformImpl, ( aOut, A, X, B ) );
    }

    void AffineTransformOp( scalar_type_t aOutputElementType, MultiTensor &aOut, scalar_value_t &A, MultiTensor &X, MemoryBuffer &B )
    {
        DISPATCH_BY_TYPE( aOutputElementType, AffineTransformImpl, ( aOut, A, X, B ) );
    }

    void AffineTransformOp( scalar_type_t aOutputElementType, MultiTensor &aOut, scalar_value_t &A, MultiTensor &X, scalar_value_t &B )
    {
        DISPATCH_BY_TYPE( aOutputElementType, AffineTransformImpl, ( aOut, A, X, B ) );
    }

    void FloorOp( MultiTensor &aOut, MultiTensor &aX )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Floor<<<lGridDim, lBlockDim>>>( aOut, aX );
    }

    void CeilOp( MultiTensor &aOut, MultiTensor &aX )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Ceil<<<lGridDim, lBlockDim>>>( aOut, aX );
    }

    template <typename _Ty>
    void AbsImpl( MultiTensor &aOut, MultiTensor &aX )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Abs<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX );
    }

    void AbsOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &aX )
    {
        DISPATCH_BY_SIGNED_TYPE( aOutputElementType, AbsImpl, ( aOut, aX ) );
    }

    template <typename _Ty>
    void SqrtImpl( MultiTensor &aOut, MultiTensor &aX )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Sqrt<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX );
    }

    void SqrtOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &aX )
    {
        DISPATCH_BY_TYPE( aOutputElementType, SqrtImpl, ( aOut, aX ) );
    }

    template <typename _Ty>
    void RoundImpl( MultiTensor &aOut, MultiTensor &aX )
    {
        int lBlockCount = ( aX.Shape().mMaxBufferSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Round<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX );
    }

    void RoundOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &aX )
    {
        DISPATCH_BY_TYPE( aOutputElementType, RoundImpl, ( aOut, aX ) );
    }

    void CountTrueOp( MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aBlockSizes, MemoryBuffer &aElementCount,
                      uint32_t aMaxBlockSize )
    {
        int lBlockCount = ( aMaxBlockSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::CountNonZero<uint8_t><<<lGridDim, lBlockDim>>>( aOut, aX, aBlockSizes, aElementCount );
    }

    template <typename _Ty>
    void CountNonZeroImpl( MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aBlockSizes, MemoryBuffer &aElementCount,
                           uint32_t aMaxBlockSize )
    {
        int lBlockCount = ( aMaxBlockSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::CountNonZero<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX, aBlockSizes, aElementCount );
    }

    void CountNonZeroOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aBlockSizes,
                         MemoryBuffer &aElementCount, uint32_t aMaxBlockSize )
    {
        DISPATCH_BY_TYPE( aOutputElementType, CountNonZeroImpl, ( aOut, aX, aBlockSizes, aElementCount, aMaxBlockSize ) );
    }

    template <typename _Ty>
    void CountZeroImpl( MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aBlockSizes, MemoryBuffer &aElementCount,
                        uint32_t aMaxBlockSize )
    {
        int lBlockCount = ( aMaxBlockSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::CountZero<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX, aBlockSizes, aElementCount );
    }

    void CountZeroOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aBlockSizes,
                      MemoryBuffer &aElementCount, uint32_t aMaxBlockSize )
    {
        DISPATCH_BY_TYPE( aOutputElementType, CountZeroImpl, ( aOut, aX, aBlockSizes, aElementCount, aMaxBlockSize ) );
    }

    template <typename _Ty>
    void ArraySummationImpl( MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aBegin, MemoryBuffer &aEnd, MemoryBuffer &aElementCount,
                             MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize )
    {
        int lBlockCount = ( aMaxBlockSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::ArraySummation<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX, aBegin, aEnd, aElementCount, aBlockSizes );
    }

    void ArraySummationOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aBegin,
                           MemoryBuffer &aEnd, MemoryBuffer &aElementCount, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize )
    {
        DISPATCH_BY_TYPE( aOutputElementType, ArraySummationImpl,
                          ( aOut, aX, aBegin, aEnd, aElementCount, aBlockSizes, aMaxBlockSize ) );
    }

    template <typename _Ty>
    void ArraySliceImpl( MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aBegin, MemoryBuffer &aEnd, MemoryBuffer &aElementCount,
                         MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize )
    {
        int lBlockCount = ( aMaxBlockSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::ArraySlice<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX, aBegin, aEnd, aElementCount, aBlockSizes );
    }

    void ArraySliceOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &aX, MemoryBuffer &aBegin, MemoryBuffer &aEnd,
                       MemoryBuffer &aElementCount, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize )
    {
        DISPATCH_BY_TYPE( aOutputElementType, ArraySliceImpl, ( aOut, aX, aBegin, aEnd, aElementCount, aBlockSizes, aMaxBlockSize ) );
    }

    template <typename _Ty>
    void DiffImpl( MultiTensor &aOut, MultiTensor &aX, uint32_t aCount, MemoryBuffer &aElementCount, MemoryBuffer &aBlockSizes,
                   uint32_t aMaxBlockSize )
    {
        int lBlockCount = ( aMaxBlockSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Diff<_Ty><<<lGridDim, lBlockDim>>>( aOut, aX, aCount, aElementCount, aBlockSizes );
    }

    void DiffOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &aX, uint32_t aCount, MemoryBuffer &aElementCount,
                 MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize )
    {
        DISPATCH_BY_TYPE( aOutputElementType, DiffImpl, ( aOut, aX, aCount, aElementCount, aBlockSizes, aMaxBlockSize ) );
    }

    template <typename _Ty>
    void ShiftImpl( MultiTensor &aOut, MultiTensor &aX, int32_t aCount, scalar_value_t &aFillValue, MemoryBuffer &aElementCount,
                    MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize )
    {
        int lBlockCount = ( aMaxBlockSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aX.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        if( aCount < 0 )
            Kernels::ShiftLeft<_Ty>
                <<<lGridDim, lBlockDim>>>( aOut, aX, -aCount, std::get<_Ty>( aFillValue ), aElementCount, aBlockSizes );
        else
            Kernels::ShiftRight<_Ty>
                <<<lGridDim, lBlockDim>>>( aOut, aX, aCount, std::get<_Ty>( aFillValue ), aElementCount, aBlockSizes );
    }

    void ShiftOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &aX, int32_t aCount, scalar_value_t &aFillValue,
                  MemoryBuffer &aElementCount, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize )
    {
        DISPATCH_BY_TYPE( aOutputElementType, ShiftImpl, ( aOut, aX, aCount, aFillValue, aElementCount, aBlockSizes, aMaxBlockSize ) );
    }

    template <typename _Ty>
    void Conv1DImpl( MultiTensor &aOut, MultiTensor &aArray0, MemoryBuffer &aElementCount0, MemoryBuffer &aBlockSizes0,
                     uint32_t aMaxElementCount0, uint32_t aMaxBlockSize0, MultiTensor &aArray1, MemoryBuffer &aElementCount1,
                     MemoryBuffer aBlockSizes1, uint32_t aMaxBlockSize1 )
    {
        int lBlockCount = ( aMaxElementCount0 / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aArray0.Shape().CountLayers(), aMaxBlockSize0, lBlockCount );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::Conv1D<_Ty>
            <<<lGridDim, lBlockDim>>>( aOut, aArray0, aElementCount0, aBlockSizes0, aArray1, aElementCount1, aBlockSizes1 );
    }

    void Conv1DOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &aArray0, MemoryBuffer &aElementCount0,
                   MemoryBuffer &aBlockSizes0, uint32_t aMaxElementCount0, uint32_t aMaxBlockSize0, MultiTensor &aArray1,
                   MemoryBuffer &aElementCount1, MemoryBuffer &aBlockSizes1, uint32_t aMaxBlockSize1 )
    {
        DISPATCH_BY_TYPE( aOutputElementType, Conv1DImpl,
                          ( aOut, aArray0, aElementCount0, aBlockSizes0, aMaxElementCount0, aMaxBlockSize0, aArray1, aElementCount1,
                            aBlockSizes1, aMaxBlockSize1 ) );
    }

    template <typename _Ty>
    void HCatImpl( MultiTensor &aOut, MultiTensor &aArray0, MemoryBuffer &aElementCount0, MultiTensor &aArray1,
                   MemoryBuffer &aElementCount1, MemoryBuffer &aBlockSizes, uint32_t aMaxBlockSize )
    {
        int lBlockCount = ( aMaxBlockSize / Private::ThreadsPerBlock ) + 1;

        dim3 lGridDim( aArray0.Shape().CountLayers(), lBlockCount, 1 );
        dim3 lBlockDim( Private::ThreadsPerBlock );

        Kernels::HCat<_Ty><<<lGridDim, lBlockDim>>>( aOut, aArray0, aElementCount0, aArray1, aElementCount1, aBlockSizes );
    }

    void HCatOp( scalar_type_t aOutputElementType, MultiTensor &aOut, MultiTensor &aArray0, MemoryBuffer &aElementCount0,
                 MultiTensor &aArray1, MemoryBuffer &aElementCount1, MemoryBuffer &aBlockSizes0, uint32_t aMaxBlockSize0 )
    {
        DISPATCH_BY_TYPE( aOutputElementType, HCatImpl,
                          ( aOut, aArray0, aElementCount0, aArray1, aElementCount1, aBlockSizes0, aMaxBlockSize0 ) );
    }

} // namespace SE::TensorOps