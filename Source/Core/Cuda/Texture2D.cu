/// @file   Texture2D.cu
///
/// @brief  Implementation file for cuda textures
///
/// @author Jean-Martin Albert
///
/// @copyright (c) 2021 LeddarTech Inc. All rights reserved.

#include "Texture2D.h"

using namespace LTSE::Core;
namespace LTSE::Cuda
{
    /// @brief Convert our internal color format into a CUDA channel description
    static hipChannelFormatDesc ToCudaChannelDesc( eColorFormat aColorFormat )
    {
        switch( aColorFormat )
        {
        case eColorFormat::R32_FLOAT:
            return hipCreateChannelDesc( 32, 0, 0, 0, hipChannelFormatKindFloat );
        case eColorFormat::RG32_FLOAT:
            return hipCreateChannelDesc( 32, 32, 0, 0, hipChannelFormatKindFloat );
        case eColorFormat::RGB32_FLOAT:
            return hipCreateChannelDesc( 32, 32, 32, 0, hipChannelFormatKindFloat );
        case eColorFormat::RGBA32_FLOAT:
            return hipCreateChannelDesc( 32, 32, 32, 32, hipChannelFormatKindFloat );
        case eColorFormat::R8_UNORM:
            return hipCreateChannelDesc( 8, 0, 0, 0, hipChannelFormatKindUnsigned );
        case eColorFormat::RG8_UNORM:
            return hipCreateChannelDesc( 8, 8, 0, 0, hipChannelFormatKindUnsigned );
        case eColorFormat::RGB8_UNORM:
            return hipCreateChannelDesc( 8, 8, 8, 0, hipChannelFormatKindUnsigned );
        case eColorFormat::RGBA8_UNORM:
        default:
            return hipCreateChannelDesc( 8, 8, 8, 8, hipChannelFormatKindUnsigned );
        }
    }

    /// @brief Convert our wrapping descriptor into a CUDA wrapping descriptor
    static hipTextureAddressMode ToCudaAddressMode( eSamplerWrapping aAddressMode )
    {
        switch( aAddressMode )
        {
        case eSamplerWrapping::REPEAT:
            return hipAddressModeWrap;
        case eSamplerWrapping::MIRRORED_REPEAT:
            return hipAddressModeMirror;
        case eSamplerWrapping::CLAMP_TO_EDGE:
            return hipAddressModeClamp;
        case eSamplerWrapping::CLAMP_TO_BORDER:
        case eSamplerWrapping::MIRROR_CLAMP_TO_BORDER:
        default:
            return hipAddressModeBorder;
        }
    }

    /// @brief Convert our filtering descriptor into a CUDA filtering descriptor
    static hipTextureFilterMode ToCudaFilterMode( eSamplerFilter aFilterMode )
    {
        switch( aFilterMode )
        {
        case eSamplerFilter::NEAREST:
            return hipFilterModePoint;
        case eSamplerFilter::LINEAR:
        default:
            return hipFilterModeLinear;
        }
    }

    Texture2D::Texture2D( sTextureCreateInfo &aSpec, std::vector<uint8_t> aData )
        : mSpec( aSpec )
    {
        hipChannelFormatDesc lTextureFormat = ToCudaChannelDesc( mSpec.mFormat );

        CUDA_ASSERT( hipMallocArray( &mInternalCudaArray, &lTextureFormat, static_cast<size_t>( mSpec.mWidth ), static_cast<size_t>( mSpec.mHeight ), hipArrayDefault ) );
        CUDA_ASSERT( hipMemcpyToArray( mInternalCudaArray, 0, 0, reinterpret_cast<void *>( aData.data() ), aData.size(), hipMemcpyHostToDevice ) );
    }

    Texture2D::Texture2D( sTextureCreateInfo &aSpec, uint8_t *aData, size_t aSize )
        : mSpec( aSpec )
    {
        hipChannelFormatDesc lTextureFormat = ToCudaChannelDesc( mSpec.mFormat );

        CUDA_ASSERT( hipMallocArray( &mInternalCudaArray, &lTextureFormat, static_cast<size_t>( mSpec.mWidth ), static_cast<size_t>( mSpec.mHeight ), hipArrayDefault ) );
        CUDA_ASSERT( hipMemcpyToArray( mInternalCudaArray, 0, 0, reinterpret_cast<void *>( aData ), aSize, hipMemcpyHostToDevice ) );
    }

    Texture2D::Texture2D( sTextureCreateInfo &aSpec, sImageData &aImageData )
        : mSpec( aSpec )
    {
        mSpec.mFormat = aImageData.mFormat;
        mSpec.mWidth  = aImageData.mWidth;
        mSpec.mHeight = aImageData.mHeight;

        hipChannelFormatDesc lTextureFormat = ToCudaChannelDesc( mSpec.mFormat );

        CUDA_ASSERT( hipMallocArray( &mInternalCudaArray, &lTextureFormat, static_cast<size_t>( mSpec.mWidth ), static_cast<size_t>( mSpec.mHeight ), hipArrayDefault ) );
        CUDA_ASSERT( hipMemcpyToArray( mInternalCudaArray, 0, 0, aImageData.mPixelData, aImageData.mByteSize, hipMemcpyHostToDevice ) );
    }

    Texture2D::~Texture2D()
    {
        if( ( nullptr != ( (void *)mInternalCudaArray ) ) )
            CUDA_ASSERT( hipFreeArray( mInternalCudaArray ) );
    }

    TextureSampler2D::TextureSampler2D( Ref<Texture2D> &aTexture, const sTextureSamplingInfo &aSamplingSpec )
        : mTexture{ aTexture }
        , mSamplingSpec{ aSamplingSpec }
    {
        hipResourceDesc lResourceDescription{};
        memset( &lResourceDescription, 0, sizeof( hipResourceDesc ) );

        lResourceDescription.resType         = hipResourceTypeArray;
        lResourceDescription.res.array.array = mTexture->mInternalCudaArray;

        hipTextureDesc lTextureDescription{};
        memset( &lTextureDescription, 0, sizeof( hipTextureDesc ) );

        lTextureDescription.readMode = hipReadModeElementType;
        if( mTexture->mSpec.mNormalizedValues )
            lTextureDescription.readMode = hipReadModeNormalizedFloat;
        lTextureDescription.borderColor[0] = mSamplingSpec.mBorderColor[0];
        lTextureDescription.borderColor[1] = mSamplingSpec.mBorderColor[1];
        lTextureDescription.borderColor[2] = mSamplingSpec.mBorderColor[2];
        lTextureDescription.borderColor[3] = mSamplingSpec.mBorderColor[3];

        lTextureDescription.addressMode[0] = ToCudaAddressMode( mSamplingSpec.mWrapping );
        lTextureDescription.addressMode[1] = ToCudaAddressMode( mSamplingSpec.mWrapping );
        lTextureDescription.addressMode[2] = ToCudaAddressMode( mSamplingSpec.mWrapping );

        lTextureDescription.filterMode = ToCudaFilterMode( mSamplingSpec.mMagnification );

        lTextureDescription.normalizedCoords = 0;
        if( mTexture->mSpec.mNormalizedCoordinates )
            lTextureDescription.normalizedCoords = 1;

        lTextureDescription.mipmapFilterMode    = hipFilterModePoint;
        lTextureDescription.mipmapLevelBias     = 0.0f;
        lTextureDescription.minMipmapLevelClamp = 0.0f;
        lTextureDescription.maxMipmapLevelClamp = 1.0f;

        mDeviceData.mScaling = math::vec2{ aSamplingSpec.mScaling[0], aSamplingSpec.mScaling[1] };
        mDeviceData.mOffset  = math::vec2{ aSamplingSpec.mOffset[0], aSamplingSpec.mOffset[1] };
        CUDA_ASSERT( hipCreateTextureObject( &( mDeviceData.mTextureObject ), &lResourceDescription, &lTextureDescription, NULL ) );
    }

} // namespace LTSE::Cuda