/// @file   TextureCubeMap.cu
///
/// @brief  Implementation file for cuda textures
///
/// @author Jean-Martin Albert
///
/// @copyright (c) 2021 LeddarTech Inc. All rights reserved.

#include "Conversion.h"
#include "TextureCubeMap.h"

using namespace SE::Core;
namespace SE::Cuda
{
    TextureCubeMap::TextureCubeMap( sTextureCreateInfo &aSpec, std::vector<uint8_t> aData )
        : mSpec( aSpec )
    {
        hipChannelFormatDesc lTextureFormat = ToCudaChannelDesc( mSpec.mFormat );

        CUDA_ASSERT( hipMallocArray( &mInternalCudaArray, &lTextureFormat, static_cast<size_t>( mSpec.mWidth ),
                                      static_cast<size_t>( mSpec.mHeight ), hipArrayDefault ) );
        CUDA_ASSERT( hipMemcpyToArray( mInternalCudaArray, 0, 0, reinterpret_cast<void *>( aData.data() ), aData.size(),
                                        hipMemcpyHostToDevice ) );
    }

    TextureCubeMap::TextureCubeMap( sTextureCreateInfo &aSpec, uint8_t *aData, size_t aSize )
        : mSpec( aSpec )
    {
        hipChannelFormatDesc lTextureFormat = ToCudaChannelDesc( mSpec.mFormat );

        CUDA_ASSERT( hipMallocArray( &mInternalCudaArray, &lTextureFormat, static_cast<size_t>( mSpec.mWidth ),
                                      static_cast<size_t>( mSpec.mHeight ), hipArrayDefault ) );
        CUDA_ASSERT( hipMemcpyToArray( mInternalCudaArray, 0, 0, reinterpret_cast<void *>( aData ), aSize, hipMemcpyHostToDevice ) );
    }

    TextureCubeMap::TextureCubeMap( sTextureCreateInfo &aSpec, sImageData &aImageData )
        : mSpec( aSpec )
    {
        mSpec.mFormat = aImageData.mFormat;
        mSpec.mWidth  = aImageData.mWidth;
        mSpec.mHeight = aImageData.mHeight;

        hipChannelFormatDesc lTextureFormat = ToCudaChannelDesc( mSpec.mFormat );

        CUDA_ASSERT( hipMallocArray( &mInternalCudaArray, &lTextureFormat, static_cast<size_t>( mSpec.mWidth ),
                                      static_cast<size_t>( mSpec.mHeight ), hipArrayDefault ) );
        CUDA_ASSERT( hipMemcpyToArray( mInternalCudaArray, 0, 0, aImageData.mPixelData.data(), aImageData.mByteSize,
                                        hipMemcpyHostToDevice ) );
    }

    TextureCubeMap::TextureCubeMap( sTextureCreateInfo &aSpec, void *aExternalBuffer, size_t aImageMemorySize )
        : mSpec( aSpec )
        , mImageMemorySize{ aImageMemorySize }
    {
        hipExternalMemoryHandleDesc lCudaExternalMemoryHandleDesc{};
        lCudaExternalMemoryHandleDesc.type                = hipExternalMemoryHandleTypeOpaqueWin32;
        lCudaExternalMemoryHandleDesc.size                = mImageMemorySize;
        lCudaExternalMemoryHandleDesc.flags               = 0;
        lCudaExternalMemoryHandleDesc.handle.win32.handle = aExternalBuffer;

        CUDA_ASSERT( hipImportExternalMemory( &mExternalMemoryHandle, &lCudaExternalMemoryHandleDesc ) );

        cudaExternalMemoryMipmappedArrayDesc lExternalMemoryMipmappedArrayDesc{};
        lExternalMemoryMipmappedArrayDesc.formatDesc = ToCudaChannelDesc( mSpec.mFormat );
        lExternalMemoryMipmappedArrayDesc.extent     = make_hipExtent( mSpec.mWidth, mSpec.mHeight, 0 );
        lExternalMemoryMipmappedArrayDesc.numLevels  = 1;
        lExternalMemoryMipmappedArrayDesc.flags      = 0;

        CUDA_ASSERT( cudaExternalMemoryGetMappedMipmappedArray( &mInternalCudaMipmappedArray, mExternalMemoryHandle,
                                                                &lExternalMemoryMipmappedArrayDesc ) );
        CUDA_ASSERT( hipGetMipmappedArrayLevel( &mInternalCudaArray, mInternalCudaMipmappedArray, 0 ) );
    }

    TextureCubeMap::~TextureCubeMap()
    {
        if( ( nullptr != ( (void *)mInternalCudaArray ) ) ) CUDA_ASSERT( hipFreeArray( mInternalCudaArray ) );
        mInternalCudaArray = nullptr;

        if( ( nullptr != ( (void *)mInternalCudaMipmappedArray ) ) )
            CUDA_ASSERT( hipFreeMipmappedArray( mInternalCudaMipmappedArray ) );
        mInternalCudaMipmappedArray = nullptr;

        if( mExternalMemoryHandle ) CUDA_ASSERT( hipDestroyExternalMemory( mExternalMemoryHandle ) );
        mExternalMemoryHandle = nullptr;
    }

    TextureSamplerCubeMap::TextureSamplerCubeMap( Ref<TextureCubeMap> &aTexture, const sTextureSamplingInfo &aSamplingSpec )
        : mTexture{ aTexture }
        , mSpec{ aSamplingSpec }
    {
        InitializeTextureSampler();
    }

    void TextureSamplerCubeMap::InitializeTextureSampler()
    {
        hipResourceDesc lResourceDescription{};
        memset( &lResourceDescription, 0, sizeof( hipResourceDesc ) );

        lResourceDescription.resType         = hipResourceTypeArray;
        lResourceDescription.res.array.array = mTexture->mInternalCudaArray;

        hipTextureDesc lTextureDescription{};
        memset( &lTextureDescription, 0, sizeof( hipTextureDesc ) );

        lTextureDescription.readMode = hipReadModeElementType;
        if( mSpec.mNormalizedValues ) lTextureDescription.readMode = hipReadModeNormalizedFloat;

        lTextureDescription.borderColor[0] = mSpec.mBorderColor[0];
        lTextureDescription.borderColor[1] = mSpec.mBorderColor[1];
        lTextureDescription.borderColor[2] = mSpec.mBorderColor[2];
        lTextureDescription.borderColor[3] = mSpec.mBorderColor[3];

        lTextureDescription.addressMode[0] = ToCudaAddressMode( mSpec.mWrapping );
        lTextureDescription.addressMode[1] = ToCudaAddressMode( mSpec.mWrapping );
        lTextureDescription.addressMode[2] = ToCudaAddressMode( mSpec.mWrapping );

        lTextureDescription.filterMode = ToCudaFilterMode( mSpec.mFilter );

        lTextureDescription.normalizedCoords = 0;
        if( mSpec.mNormalizedCoordinates ) lTextureDescription.normalizedCoords = 1;

        lTextureDescription.mipmapFilterMode    = hipFilterModePoint;
        lTextureDescription.mipmapLevelBias     = 0.0f;
        lTextureDescription.minMipmapLevelClamp = 0.0f;
        lTextureDescription.maxMipmapLevelClamp = 1.0f;

        mDeviceData.mScaling = math::vec2{ mSpec.mScaling[0], mSpec.mScaling[1] };
        mDeviceData.mOffset  = math::vec2{ mSpec.mOffset[0], mSpec.mOffset[1] };
        CUDA_ASSERT( hipCreateTextureObject( &( mDeviceData.mTextureObject ), &lResourceDescription, &lTextureDescription, NULL ) );
    }

} // namespace SE::Cuda