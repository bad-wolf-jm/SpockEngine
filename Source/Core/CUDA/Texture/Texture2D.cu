/// @file   Texture2D.cu
///
/// @brief  Implementation file for cuda textures
///
/// @author Jean-Martin Albert
///
/// @copyright (c) 2021 LeddarTech Inc. All rights reserved.

#include "Texture2D.h"

using namespace SE::Core;
namespace SE::Cuda
{
    /// @brief Convert our internal color format into a CUDA channel description
    hipChannelFormatDesc ToCudaChannelDesc( eColorFormat aColorFormat )
    {
        switch( aColorFormat )
        {
        case eColorFormat::R32_FLOAT: return hipCreateChannelDesc( 32, 0, 0, 0, hipChannelFormatKindFloat );
        case eColorFormat::RG32_FLOAT: return hipCreateChannelDesc( 32, 32, 0, 0, hipChannelFormatKindFloat );
        case eColorFormat::RGB32_FLOAT: return hipCreateChannelDesc( 32, 32, 32, 0, hipChannelFormatKindFloat );
        case eColorFormat::RGBA32_FLOAT: return hipCreateChannelDesc( 32, 32, 32, 32, hipChannelFormatKindFloat );
        case eColorFormat::R8_UNORM: return hipCreateChannelDesc( 8, 0, 0, 0, hipChannelFormatKindUnsigned );
        case eColorFormat::RG8_UNORM: return hipCreateChannelDesc( 8, 8, 0, 0, hipChannelFormatKindUnsigned );
        case eColorFormat::RGB8_UNORM: return hipCreateChannelDesc( 8, 8, 8, 0, hipChannelFormatKindUnsigned );
        case eColorFormat::RGBA8_UNORM:
        default: return hipCreateChannelDesc( 8, 8, 8, 8, hipChannelFormatKindUnsigned );
        }
    }

    /// @brief Convert our wrapping descriptor into a CUDA wrapping descriptor
    static hipTextureAddressMode ToCudaAddressMode( eSamplerWrapping aAddressMode )
    {
        switch( aAddressMode )
        {
        case eSamplerWrapping::REPEAT: return hipAddressModeWrap;
        case eSamplerWrapping::MIRRORED_REPEAT: return hipAddressModeMirror;
        case eSamplerWrapping::CLAMP_TO_EDGE: return hipAddressModeClamp;
        case eSamplerWrapping::CLAMP_TO_BORDER:
        case eSamplerWrapping::MIRROR_CLAMP_TO_BORDER:
        default: return hipAddressModeBorder;
        }
    }

    /// @brief Convert our filtering descriptor into a CUDA filtering descriptor
    static hipTextureFilterMode ToCudaFilterMode( eSamplerFilter aFilterMode )
    {
        switch( aFilterMode )
        {
        case eSamplerFilter::NEAREST: return hipFilterModePoint;
        case eSamplerFilter::LINEAR:
        default: return hipFilterModeLinear;
        }
    }

    Texture2D::Texture2D( sTextureCreateInfo &aSpec, std::vector<uint8_t> aData )
        : mSpec( aSpec )
    {
        hipChannelFormatDesc lTextureFormat = ToCudaChannelDesc( mSpec.mFormat );

        CUDA_ASSERT( hipMallocArray( &mInternalCudaArray, &lTextureFormat, static_cast<size_t>( mSpec.mWidth ),
                                      static_cast<size_t>( mSpec.mHeight ), hipArrayDefault ) );
        CUDA_ASSERT( hipMemcpyToArray( mInternalCudaArray, 0, 0, reinterpret_cast<void *>( aData.data() ), aData.size(),
                                        hipMemcpyHostToDevice ) );
    }

    Texture2D::Texture2D( sTextureCreateInfo &aSpec, uint8_t *aData, size_t aSize )
        : mSpec( aSpec )
    {
        hipChannelFormatDesc lTextureFormat = ToCudaChannelDesc( mSpec.mFormat );

        CUDA_ASSERT( hipMallocArray( &mInternalCudaArray, &lTextureFormat, static_cast<size_t>( mSpec.mWidth ),
                                      static_cast<size_t>( mSpec.mHeight ), hipArrayDefault ) );
        CUDA_ASSERT( hipMemcpyToArray( mInternalCudaArray, 0, 0, reinterpret_cast<void *>( aData ), aSize, hipMemcpyHostToDevice ) );
    }

    Texture2D::Texture2D( sTextureCreateInfo &aSpec, sImageData &aImageData )
        : mSpec( aSpec )
    {
        mSpec.mFormat = aImageData.mFormat;
        mSpec.mWidth  = aImageData.mWidth;
        mSpec.mHeight = aImageData.mHeight;

        hipChannelFormatDesc lTextureFormat = ToCudaChannelDesc( mSpec.mFormat );

        CUDA_ASSERT( hipMallocArray( &mInternalCudaArray, &lTextureFormat, static_cast<size_t>( mSpec.mWidth ),
                                      static_cast<size_t>( mSpec.mHeight ), hipArrayDefault ) );
        CUDA_ASSERT(
            hipMemcpyToArray( mInternalCudaArray, 0, 0, aImageData.mPixelData.data(), aImageData.mByteSize, hipMemcpyHostToDevice ) );
    }

    Texture2D::Texture2D( sTextureCreateInfo &aSpec, void *aExternalBuffer, size_t aImageMemorySize )
        : mSpec( aSpec )
        , mImageMemorySize{ aImageMemorySize }
    {
        hipExternalMemoryHandleDesc lCudaExternalMemoryHandleDesc{};
        lCudaExternalMemoryHandleDesc.type                = hipExternalMemoryHandleTypeOpaqueWin32;
        lCudaExternalMemoryHandleDesc.size                = mImageMemorySize;
        lCudaExternalMemoryHandleDesc.flags               = 0;
        lCudaExternalMemoryHandleDesc.handle.win32.handle = aExternalBuffer;

        CUDA_ASSERT( hipImportExternalMemory( &mExternalMemoryHandle, &lCudaExternalMemoryHandleDesc ) );

        cudaExternalMemoryMipmappedArrayDesc lExternalMemoryMipmappedArrayDesc{};
        lExternalMemoryMipmappedArrayDesc.formatDesc = ToCudaChannelDesc( mSpec.mFormat );
        lExternalMemoryMipmappedArrayDesc.extent     = make_hipExtent( mSpec.mWidth, mSpec.mHeight, 0 );
        lExternalMemoryMipmappedArrayDesc.numLevels  = 1;
        lExternalMemoryMipmappedArrayDesc.flags      = 0;

        CUDA_ASSERT( cudaExternalMemoryGetMappedMipmappedArray( &mInternalCudaMipmappedArray, mExternalMemoryHandle,
                                                                &lExternalMemoryMipmappedArrayDesc ) );
        CUDA_ASSERT( hipGetMipmappedArrayLevel( &mInternalCudaArray, mInternalCudaMipmappedArray, 0 ) );
    }

    Texture2D::~Texture2D()
    {
        if( ( nullptr != ( (void *)mInternalCudaArray ) ) ) CUDA_ASSERT( hipFreeArray( mInternalCudaArray ) );
        mInternalCudaArray = nullptr;

        if( ( nullptr != ( (void *)mInternalCudaMipmappedArray ) ) )
            CUDA_ASSERT( hipFreeMipmappedArray( mInternalCudaMipmappedArray ) );
        mInternalCudaMipmappedArray = nullptr;

        if( mExternalMemoryHandle ) CUDA_ASSERT( hipDestroyExternalMemory( mExternalMemoryHandle ) );
        mExternalMemoryHandle = nullptr;
    }

    TextureSampler2D::TextureSampler2D( Ref<Texture2D> &aTexture, const sTextureSamplingInfo &aSamplingSpec )
        : mTexture{ aTexture }
        , mSpec{ aSamplingSpec }
    {
        InitializeTextureSampler();
    }

    void TextureSampler2D::InitializeTextureSampler()
    {
        hipResourceDesc lResourceDescription{};
        memset( &lResourceDescription, 0, sizeof( hipResourceDesc ) );

        lResourceDescription.resType         = hipResourceTypeArray;
        lResourceDescription.res.array.array = mTexture->mInternalCudaArray;

        hipTextureDesc lTextureDescription{};
        memset( &lTextureDescription, 0, sizeof( hipTextureDesc ) );

        lTextureDescription.readMode = hipReadModeElementType;
        if( mSpec.mNormalizedValues ) lTextureDescription.readMode = hipReadModeNormalizedFloat;
        lTextureDescription.borderColor[0] = mSpec.mBorderColor[0];
        lTextureDescription.borderColor[1] = mSpec.mBorderColor[1];
        lTextureDescription.borderColor[2] = mSpec.mBorderColor[2];
        lTextureDescription.borderColor[3] = mSpec.mBorderColor[3];

        lTextureDescription.addressMode[0] = ToCudaAddressMode( mSpec.mWrapping );
        lTextureDescription.addressMode[1] = ToCudaAddressMode( mSpec.mWrapping );
        lTextureDescription.addressMode[2] = ToCudaAddressMode( mSpec.mWrapping );

        lTextureDescription.filterMode = ToCudaFilterMode( mSpec.mFilter );

        lTextureDescription.normalizedCoords = 0;
        if( mSpec.mNormalizedCoordinates ) lTextureDescription.normalizedCoords = 1;

        lTextureDescription.mipmapFilterMode    = hipFilterModePoint;
        lTextureDescription.mipmapLevelBias     = 0.0f;
        lTextureDescription.minMipmapLevelClamp = 0.0f;
        lTextureDescription.maxMipmapLevelClamp = 1.0f;

        mDeviceData.mScaling = math::vec2{ mSpec.mScaling[0], mSpec.mScaling[1] };
        mDeviceData.mOffset  = math::vec2{ mSpec.mOffset[0], mSpec.mOffset[1] };
        CUDA_ASSERT( hipCreateTextureObject( &( mDeviceData.mTextureObject ), &lResourceDescription, &lTextureDescription, NULL ) );
    }

} // namespace SE::Cuda