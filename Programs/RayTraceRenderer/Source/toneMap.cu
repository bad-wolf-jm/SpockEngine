#include "hip/hip_runtime.h"
#include "Core/GPUResource/CudaAssert.h"

#include "SampleRenderer.h"

using namespace osc;

namespace osc
{

    inline SE_CUDA_DEVICE_FUNCTION_DEF float4 sqrt( float4 f )
    {
        return make_float4( sqrtf( f.x ), sqrtf( f.y ), sqrtf( f.z ), sqrtf( f.w ) );
    }
    inline SE_CUDA_DEVICE_FUNCTION_DEF float  clampf( float f ) { return min( 1.f, max( 0.f, f ) ); }
    inline SE_CUDA_DEVICE_FUNCTION_DEF float4 clamp( float4 f )
    {
        return make_float4( clampf( f.x ), clampf( f.y ), clampf( f.z ), clampf( f.w ) );
    }

    /*! runs a cuda kernel that performs gamma correction and float4-to-rgba conversion */
    CUDA_KERNEL_DEFINITION void computeFinalPixelColorsKernel( uint32_t *finalColorBuffer, float4 *denoisedBuffer, math::ivec2 size )
    {
        int pixelX = threadIdx.x + blockIdx.x * blockDim.x;
        int pixelY = threadIdx.y + blockIdx.y * blockDim.y;
        if( pixelX >= size.x ) return;
        if( pixelY >= size.y ) return;

        int pixelID = pixelX + size.x * pixelY;

        float4 f4     = denoisedBuffer[pixelID];
        f4            = clamp( sqrt( f4 ) );
        uint32_t rgba = 0;
        rgba |= (uint32_t)( f4.x * 255.9f ) << 0;
        rgba |= (uint32_t)( f4.y * 255.9f ) << 8;
        rgba |= (uint32_t)( f4.z * 255.9f ) << 16;
        rgba |= (uint32_t)255 << 24;
        finalColorBuffer[pixelID] = rgba;
    }

    void SampleRenderer::computeFinalPixelColors()
    {
        math::ivec2 fbSize = launchParams.mFrame.mSize;
        math::ivec2 blockSize( 32 );
        math::ivec2 numBlocks = divRoundUp( fbSize, blockSize );
        computeFinalPixelColorsKernel<<<dim3( numBlocks.x, numBlocks.y ), dim3( blockSize.x, blockSize.y )>>>(
            (uint32_t *)finalColorBuffer.RawDevicePtr(), (float4 *)denoisedBuffer.RawDevicePtr(), fbSize );
    }

} // namespace osc
